#include <assert.h>
#include <stdio.h>
#include "mvt_kernel.hu"
/* mvt.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>
#include <timer.hpp>
#include <texture.hu>

/* Include benchmark-specific header. */
#include "mvt.h"

#if defined(__HIPCC__) && !defined(CUDA_DEVICE)
#define CUDA_DEVICE 0
#endif

/* Array initialization. */
static
void init_array(int n,
		DATA_TYPE POLYBENCH_1D(x1,N,n),
		DATA_TYPE POLYBENCH_1D(x2,N,n),
		DATA_TYPE POLYBENCH_1D(y_1,N,n),
		DATA_TYPE POLYBENCH_1D(y_2,N,n),
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j;

  for (i = 0; i < n; i++)
    {
      x1[i] = (DATA_TYPE) (i % n) / n;
      x2[i] = (DATA_TYPE) ((i + 1) % n) / n;
      y_1[i] = (DATA_TYPE) ((i + 3) % n) / n;
      y_2[i] = (DATA_TYPE) ((i + 4) % n) / n;
      for (j = 0; j < n; j++)
	A[i][j] = (DATA_TYPE) (i*j % n) / n;
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(x1,N,n),
		 DATA_TYPE POLYBENCH_1D(x2,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("x1");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, x1[i]);
  }
  POLYBENCH_DUMP_END("x1");

  POLYBENCH_DUMP_BEGIN("x2");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, x2[i]);
  }
  POLYBENCH_DUMP_END("x2");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_mvt(int n,
		DATA_TYPE POLYBENCH_1D(x1,N,n),
		DATA_TYPE POLYBENCH_1D(x2,N,n),
		DATA_TYPE POLYBENCH_1D(y_1,N,n),
		DATA_TYPE POLYBENCH_1D(y_2,N,n),
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j;

  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    TextureObject_t tex_A;
    createTextureObject(&tex_A, A, 2000, 2000);
    float *dev_x1;
    float *dev_x2;
    float *dev_y_1;
    float *dev_y_2;
    
    cudaCheckReturn(hipMalloc((void **) &dev_x1, (2000) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_x2, (2000) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_y_1, (2000) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_y_2, (2000) * sizeof(float)));
    
    cudaCheckReturn(hipMemcpy(dev_x1, x1, (2000) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x2, x2, (2000) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_y_1, y_1, (2000) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_y_2, y_2, (2000) * sizeof(float), hipMemcpyHostToDevice));

    GpuEventTimer gpu_timer;
    gpu_timer.Start();
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(63);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (tex_A.tex, dev_x1, dev_y_1);
      cudaCheckKernel();
    }
    gpu_timer.Stop();
    printf("kernel0: %lf\n", gpu_timer.ElapsedTime<TimeUnit::S>());
    
    gpu_timer.Start();
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(63);
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (tex_A.tex, dev_x2, dev_y_2);
      cudaCheckKernel();
    }
    gpu_timer.Stop();
    printf("kernel1: %lf\n", gpu_timer.ElapsedTime<TimeUnit::S>());
    
    cudaCheckReturn(hipMemcpy(x1, dev_x1, (2000) * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(x2, dev_x2, (2000) * sizeof(float), hipMemcpyDeviceToHost));
    destroyTextureObject(&tex_A);
    cudaCheckReturn(hipFree(dev_x1));
    cudaCheckReturn(hipFree(dev_x2));
    cudaCheckReturn(hipFree(dev_y_1));
    cudaCheckReturn(hipFree(dev_y_2));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_1D_ARRAY_DECL(x1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(x2, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y_1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y_2, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n,
	      POLYBENCH_ARRAY(x1),
	      POLYBENCH_ARRAY(x2),
	      POLYBENCH_ARRAY(y_1),
	      POLYBENCH_ARRAY(y_2),
	      POLYBENCH_ARRAY(A));

#if defined(__HIPCC__) && defined(POLYBENCH_TIME_NO_CUDA_INIT_CTX)
  hipSetDevice(CUDA_DEVICE);
#endif

  /* Start timer. */
  polybench_start_instruments;

#if defined(__HIPCC__) && !defined(POLYBENCH_TIME_NO_CUDA_INIT_CTX)
  hipSetDevice(CUDA_DEVICE);
#endif

  /* Run kernel. */
  kernel_mvt (n,
	      POLYBENCH_ARRAY(x1),
	      POLYBENCH_ARRAY(x2),
	      POLYBENCH_ARRAY(y_1),
	      POLYBENCH_ARRAY(y_2),
	      POLYBENCH_ARRAY(A));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(x1), POLYBENCH_ARRAY(x2)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(x1);
  POLYBENCH_FREE_ARRAY(x2);
  POLYBENCH_FREE_ARRAY(y_1);
  POLYBENCH_FREE_ARRAY(y_2);

  return 0;
}
